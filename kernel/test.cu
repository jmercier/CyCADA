
#include <hip/hip_runtime.h>
extern "C"
__global__ void fill(unsigned int *a, unsigned int value, unsigned int size)
{
    const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= size)
        return;

    a[index] = value;
}
